/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication and is exactly the same as
 * Chapter 7 of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <error_checker.h>
// includes, kernels
#include <matrixMul_kernel.cuh>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

#define cutilSafeCallNoSync(err)     __cudaSafeCallNoSync(err, __FILE__, __LINE__)
#define cutilSafeCall(err)           __cudaSafeCall      (err, __FILE__, __LINE__)
#define cutilSafeThreadSync()        __cudaSafeThreadSync(__FILE__, __LINE__)
#define cutilCheckError(err)         __cutilCheckError   (err, __FILE__, __LINE__)
#define cutilCheckMsg(msg)           __cutilGetLastError (msg, __FILE__, __LINE__)
#define cutilCheckMsgAndSync(msg)    __cutilGetLastErrorAndSync (msg, __FILE__, __LINE__)
#define cutilSafeMalloc(mallocCall)  __cutilSafeMalloc   ((mallocCall), __FILE__, __LINE__)
#define cutilCondition(val)          __cutilCondition    (val, __FILE__, __LINE__)
#define cutilExit(argc, argv)        __cutilExit         (argc, argv)

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
   if( hipSuccess != err) {
      fprintf(stderr, "%s(%i) : cudaSafeCall() Runtime API error %d: %s.\n",
               file, line, (int)err, hipGetErrorString( err ) );
      exit(-1);
   }
}

inline void __cutilCheckError( int err, const char *file, const int line )
{
   if( err != 0 ) {
      fprintf(stderr, "%s(%i) : CUTIL CUDA error.\n",
              file, line);
      exit(-1);
   }
}

inline void __cutilGetLastError( const char *errorMessage, const char *file, const int line )
{
   hipError_t err = hipGetLastError();
   if( hipSuccess != err) {
      fprintf(stderr, "%s(%i) : cutilCheckMsg() CUTIL CUDA error : %s : (%d) %s.\n",
              file, line, errorMessage, (int)err, hipGetErrorString( err ) );
      exit(-1);
   }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int
main(int argc, char** argv)
{
    runTest(argc, argv);

    exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{
    hipSetDevice( gpuGetMaxGflopsDeviceId() );

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    cutilSafeCall(hipMalloc((void**) &d_A, mem_size_A));
    float* d_B;
    cutilSafeCall(hipMalloc((void**) &d_B, mem_size_B));

    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    cutilSafeCall(hipMalloc((void**) &d_C, mem_size_C));

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    
    // create and start timer
//    unsigned int timer = 0;
    StopWatchInterface *timer = NULL; // timer object
    cutilCheckError(sdkCreateTimer(&timer));
    cutilCheckError(sdkStartTimer(&timer));

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);

    // execute the kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );

    // stop and destroy timer
    cutilCheckError(sdkStopTimer(&timer));
    printf("Processing time: %f (ms) \n", sdkGetTimerValue(&timer));
    cutilCheckError(sdkDeleteTimer(&timer));

    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    computeGold(reference, h_A, h_B, HA, WA, WB);

    // check result
//    int res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
//    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
//    if (res!=1) printDiff(reference, h_C, WC, HC);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    cutilSafeCall(hipFree(d_A));
    cutilSafeCall(hipFree(d_B));
    cutilSafeCall(hipFree(d_C));

    hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (data1[k] != data2[k]) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}

